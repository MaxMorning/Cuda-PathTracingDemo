﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define RENDER_WIDTH 512
#define RENDER_HEIGHT 512
#define TILE_SIZE 16
#define STACK_CAPACITY 128
#define SHARED_MEM_CAP STACK_CAPACITY * RENDER_WIDTH * RENDER_HEIGHT
#define SPP 1024
#define RR_RATE 0.9
#define PI 3.1415926

// BMP Operation
// 文件信息头结构体
typedef struct
{
    unsigned int   bfSize;        // 文件大小 以字节为单位(2-5字节)
    unsigned short bfReserved1;   // 保留，必须设置为0 (6-7字节)
    unsigned short bfReserved2;   // 保留，必须设置为0 (8-9字节)
    unsigned int   bfOffBits;     // 从文件头到像素数据的偏移  (10-13字节)
} _BITMAPFILEHEADER;

//图像信息头结构体
typedef struct
{
    unsigned int    biSize;          // 此结构体的大小 (14-17字节)
    int             biWidth;         // 图像的宽  (18-21字节)
    int             biHeight;        // 图像的高  (22-25字节)
    unsigned short  biPlanes;        // 表示bmp图片的平面属，显然显示器只有一个平面，所以恒等于1 (26-27字节)
    unsigned short  biBitCount;      // 一像素所占的位数，一般为24   (28-29字节)
    unsigned int    biCompression;   // 说明图象数据压缩的类型，0为不压缩。 (30-33字节)
    unsigned int    biSizeImage;     // 像素数据所占大小, 这个值应该等于上面文件头结构中bfSize-bfOffBits (34-37字节)
    int             biXPelsPerMeter; // 说明水平分辨率，用象素/米表示。一般为0 (38-41字节)
    int             biYPelsPerMeter; // 说明垂直分辨率，用象素/米表示。一般为0 (42-45字节)
    unsigned int    biClrUsed;       // 说明位图实际使用的彩色表中的颜色索引数（设为0的话，则说明使用所有调色板项）。 (46-49字节)
    unsigned int    biClrImportant;  // 说明对图象显示有重要影响的颜色索引的数目，如果是0，表示都重要。(50-53字节)
} _BITMAPINFOHEADER;

__host__ void save_image(unsigned char* target_img, int width, int height)
{
    FILE* file_ptr = fopen("RenderResult.bmp", "wb+");

    unsigned short fileType = 0x4d42;
    _BITMAPFILEHEADER fileHeader;
    _BITMAPINFOHEADER infoHeader;

    fileHeader.bfSize = (width) * (height) * 3 + 54;
    fileHeader.bfReserved1 = 0;
    fileHeader.bfReserved2 = 0;
    fileHeader.bfOffBits = 54;

    infoHeader.biSize = 40;
    infoHeader.biWidth = width;
    infoHeader.biHeight = height;
    infoHeader.biPlanes = 1;
    infoHeader.biBitCount = 24;
    infoHeader.biCompression = 0;
    infoHeader.biSizeImage = (width) * (height) * 3;
    infoHeader.biXPelsPerMeter = 0;
    infoHeader.biYPelsPerMeter = 0;
    infoHeader.biClrUsed = 0;
    infoHeader.biClrImportant = 0;

    fwrite(&fileType, sizeof(unsigned short), 1, file_ptr);
    fwrite(&fileHeader, sizeof(_BITMAPFILEHEADER), 1, file_ptr);
    fwrite(&infoHeader, sizeof(_BITMAPINFOHEADER), 1, file_ptr);

    fwrite(target_img, sizeof(unsigned char), (height) * (width) * 3, file_ptr);

    fclose(file_ptr);
}

// 3D resources

struct Trianle {
    float3 tri_a;
    float3 tri_b;
    float3 tri_c;
    float3 normal_line;
    bool is_light;
    float brdf_rate;
};

/*
// test scene
// Light triagles
#define LIGHT_TRI_COUNT 2

__constant__ float d_light_irradiance = 40;

// object triagles
// No BVH
#define BRDF_rate 0.6

#define OBJ_TRI_COUNT 24
Trianle h_scene_objects[] = {
    // light tri
    Trianle{float3{110, 110, 300}, float3{110, 190, 300}, float3{190, 110, 300}, float3{0, 0, -1}, true, BRDF_rate},
    Trianle{float3{190, 110, 300}, float3{110, 190, 300}, float3{190, 190, 300}, float3{0, 0, -1}, true, BRDF_rate},
    // Trianle{float3{110, 110, 301}, float3{110, 190, 301}, float3{190, 110, 301}, float3{0, 0, 1}, true, BRDF_rate},
    // Trianle{float3{190, 110, 301}, float3{110, 190, 301}, float3{190, 190, 301}, float3{0, 0, 1}, true, BRDF_rate},
    // internal box 100 * 100 * 30
    // top
    Trianle{float3{100, 100, 100}, float3{200, 100, 100}, float3{100, 200, 100}, float3{0, 0, 1}, false, BRDF_rate},
    Trianle{float3{200, 100, 100}, float3{200, 200, 100}, float3{100, 200, 100}, float3{0, 0, 1}, false, BRDF_rate},
    // bottom
    Trianle{float3{100, 100, 70}, float3{200, 100, 70}, float3{100, 200, 70}, float3{0, 0, -1}, false, BRDF_rate},
    Trianle{float3{200, 100, 70}, float3{200, 200, 70}, float3{100, 200, 70}, float3{0, 0, -1}, false, BRDF_rate},
    // front
    Trianle{float3{100, 100, 100}, float3{200, 100, 100}, float3{100, 100, 70}, float3{0, -1, 0}, false, BRDF_rate},
    Trianle{float3{100, 100, 70}, float3{200, 100, 70}, float3{200, 100, 100}, float3{0, -1, 0}, false, BRDF_rate},

    // behind
    Trianle{float3{100, 200, 100}, float3{200, 200, 100}, float3{100, 200, 70}, float3{0, 1, 0}, false, BRDF_rate},
    Trianle{float3{100, 200, 70}, float3{200, 200, 70}, float3{200, 200, 100}, float3{0, 1, 0}, false, BRDF_rate},

    // left
    Trianle{float3{100, 100, 100}, float3{100, 200, 100}, float3{100, 100, 70}, float3{-1, 0, 0}, false, BRDF_rate},
    Trianle{float3{100, 100, 70}, float3{100, 200, 70}, float3{100, 200, 100}, float3{-1, 0, 0}, false, BRDF_rate},

    // right
    Trianle{float3{200, 100, 100}, float3{200, 200, 100}, float3{200, 100, 70}, float3{1, 0, 0}, false, BRDF_rate},
    Trianle{float3{200, 100, 70}, float3{200, 200, 70}, float3{200, 200, 100}, float3{1, 0, 0}, false, BRDF_rate},

    // general box 300 * 300 * 300.001
    // top
    Trianle{float3{0, 0, 300.001}, float3{0, 300, 300.001}, float3{300, 0, 300.001}, float3{0, 0, -1}, false, BRDF_rate},
    Trianle{float3{0, 300, 300.001}, float3{300, 0, 300.001}, float3{300, 300, 300.001}, float3{0, 0, -1}, false, BRDF_rate},

    // bottom
    Trianle{float3{0, 0, 0}, float3{0, 300, 0}, float3{300, 0, 0}, float3{0, 0, 1}, false, BRDF_rate},
    Trianle{float3{0, 300, 0}, float3{300, 0, 0}, float3{300, 300, 0}, float3{0, 0, 1}, false, BRDF_rate},

    // left
    Trianle{float3{0, 0, 0}, float3{0, 0, 300.001}, float3{0, 300, 300.001}, float3{1, 0, 0}, false, BRDF_rate},
    Trianle{float3{0, 300, 300.001}, float3{0, 300, 0}, float3{0, 0, 0}, float3{1, 0, 0}, false, BRDF_rate},

    // right
    Trianle{float3{300, 0, 0}, float3{300, 0, 300.001}, float3{300, 300, 300.001}, float3{-1, 0, 0}, false, BRDF_rate},
    Trianle{float3{300, 300, 300.001}, float3{300, 300, 0}, float3{300, 0, 0}, float3{-1, 0, 0}, false, BRDF_rate},

    // behind
    Trianle{float3{0, 300, 0}, float3{0, 300, 300.001}, float3{300, 300, 0}, float3{0, -1, 0}, false, BRDF_rate},
    Trianle{float3{300, 300, 0}, float3{300, 300, 300.001}, float3{0, 300, 300.001}, float3{0, -1, 0}, false, BRDF_rate}
};

__constant__ Trianle d_scene_objects[OBJ_TRI_COUNT];


// camera position
__constant__ float3 d_camera_position = float3{150, -400, 150};
__constant__ float3 d_camera_direction = float3{0, 1, 0};
__constant__ float3 d_camera_up_direction = float3{0, 0, 1};
__constant__ float3 d_camera_left_direction = float3{1, 0, 0};
// 浮点精度考虑，设置较大焦距和成像平面
__constant__ float d_camera_focal_length = 200;
__constant__ float d_camera_width = 150;
__constant__ float d_camera_height = 150;
__constant__ float d_camera_pixel_width = 150.0 / RENDER_WIDTH;
__constant__ float d_camera_pixel_height= 150.0 / RENDER_HEIGHT;
*/



// Cornell box
#define LIGHT_TRI_COUNT 2
__constant__ float d_light_irradiance = 42;

#define BRDF_rate 0.74
#define OBJ_TRI_COUNT 32
// Trianle{float3{}, float3{}, float3{}, float3{}, false, BRDF_rate},
Trianle h_scene_objects[] = {
    // Light triagles
    Trianle{float3{343.0, 548.799, 227.0}, float3{343.0, 548.799, 332.0}, float3{213.0, 548.799, 332.0}, float3{0, -1, 0}, true, BRDF_rate},
    Trianle{float3{343.0, 548.799, 227.0}, float3{213.0, 548.799, 227.0}, float3{213.0, 548.799, 332.0}, float3{0, -1, 0}, true, BRDF_rate},
    
    // Floor
    Trianle{float3{552.8, 0.0, 0.0}, float3{0.0, 0.0, 0.0}, float3{0.0, 0.0, 559.2}, float3{0, 1, 0}, false, BRDF_rate},
    Trianle{float3{552.8, 0.0, 0.0}, float3{549.6, 0.0, 559.2}, float3{0.0, 0.0, 559.2}, float3{0, 1, 0}, false, BRDF_rate},

    // Ceiling
    Trianle{float3{556.0, 548.8, 0.0}, float3{556.0, 548.8, 559.2}, float3{0.0, 548.8, 559.2}, float3{0, -1, 0}, false, BRDF_rate},
    Trianle{float3{556.0, 548.8, 0.0}, float3{0.0, 548.8, 0.0}, float3{0.0, 548.8, 559.2}, float3{0, -1, 0}, false, BRDF_rate},

    // Back wall
    Trianle{float3{549.6, 0.0, 559.2}, float3{0.0, 0.0, 559.2}, float3{0.0, 548.8, 559.2}, float3{0, 0, -1}, false, BRDF_rate},
    Trianle{float3{549.6, 0.0, 559.2}, float3{556.0, 548.8, 559.2}, float3{0.0, 548.8, 559.2}, float3{0, 0, -1}, false, BRDF_rate},

    // Right wall
    Trianle{float3{0.0, 0.0, 559.2}, float3{0.0, 0.0, 0.0}, float3{0.0, 548.8, 0.0}, float3{1, 0, 0}, false, BRDF_rate},
    Trianle{float3{0.0, 0.0, 559.2}, float3{0.0, 548.8, 559.2}, float3{0.0, 548.8, 0.0}, float3{1, 0, 0}, false, BRDF_rate},

    // Left wall
    Trianle{float3{552.8, 0.0, 0.0}, float3{549.6, 0.0, 559.2}, float3{556.0, 548.8, 559.2}, float3{-1, 0, 0}, false, BRDF_rate},
    Trianle{float3{552.8, 0.0, 0.0}, float3{556.0, 548.8, 0.0}, float3{556.0, 548.8, 559.2}, float3{-1, 0, 0}, false, BRDF_rate},
    
    // Short block
    // Top
    Trianle{float3{130.0, 165.0, 65.0}, float3{82.0, 165.0, 225.0}, float3{240.0, 165.0, 272.0}, float3{0, 1, 0}, false, BRDF_rate},
    Trianle{float3{130.0, 165.0, 65.0}, float3{290.0, 165.0, 114.0}, float3{240.0, 165.0, 272.0}, float3{0, 1, 0}, false, BRDF_rate},
    
    // Left
    Trianle{float3{290.0, 0.0, 114.0}, float3{290.0, 165.0, 114.0}, float3{240.0, 165.0, 272.0}, float3{-0.9534, 0, -0.301709}, false, BRDF_rate},
    Trianle{float3{290.0, 0.0, 114.0}, float3{240.0, 0.0, 272.0}, float3{240.0, 165.0, 272.0}, float3{-0.9534, 0, -0.301709}, false, BRDF_rate},
    
    // Front
    Trianle{float3{130.0, 0.0, 65.0}, float3{130.0, 165.0, 65.0}, float3{290.0, 165.0, 114.0}, float3{-0.292826, 0, -0.956166}, false, BRDF_rate},
    Trianle{float3{130.0, 0.0, 65.0}, float3{290.0, 0.0, 114.0}, float3{290.0, 165.0, 114.0}, float3{-0.292826, 0, -0.956166}, false, BRDF_rate},

    // Right
    Trianle{float3{82.0, 0.0, 225.0}, float3{82.0, 165.0, 225.0}, float3{130.0, 165.0, 65.0}, float3{-0.957826, 0, -0.287348}, false, BRDF_rate},
    Trianle{float3{82.0, 0.0, 225.0}, float3{130.0, 0.0, 65.0}, float3{130.0, 165.0, 65.0}, float3{-0.957826, 0, -0.287348}, false, BRDF_rate},
    
    // Behind
    Trianle{float3{240.0, 0.0, 272.0}, float3{240.0, 165.0, 272.0}, float3{82.0, 165.0, 225.0}, float3{-0.285121, 0, -0.958492}, false, BRDF_rate},
    Trianle{float3{240.0, 0.0, 272.0}, float3{82.0, 0.0, 225.0}, float3{82.0, 165.0, 225.0}, float3{-0.285121, 0, -0.958492}, false, BRDF_rate},

    // Tall block
    // Top
    Trianle{float3{423.0, 330.0, 247.0}, float3{265.0, 330.0, 296.0}, float3{314.0, 330.0, 456.0}, float3{0, 1, 0}, false, BRDF_rate},
    Trianle{float3{423.0, 330.0, 247.0}, float3{472.0, 330.0, 406.0}, float3{314.0, 330.0, 456.0}, float3{0, 1, 0}, false, BRDF_rate},

    // Left
    Trianle{float3{423.0, 0.0, 247.0}, float3{423.0, 330.0, 247.0}, float3{472.0, 330.0, 406.0}, float3{0.955649, 0, -0.294508}, false, BRDF_rate},
    Trianle{float3{423.0, 0.0, 247.0}, float3{472.0, 0.0, 406.0}, float3{472.0, 330.0, 406.0}, float3{0.955649, 0, -0.294508}, false, BRDF_rate},

    // Behind
    Trianle{float3{472.0, 0.0, 406.0}, float3{472.0, 330.0, 406.0}, float3{314.0, 330.0, 456.0}, float3{-0.301709, 0, -0.953400}, false, BRDF_rate},
    Trianle{float3{472.0, 0.0, 406.0}, float3{314.0, 0.0, 456.0}, float3{314.0, 330.0, 456.0}, float3{-0.301709, 0, -0.953400}, false, BRDF_rate},

    // Right
    Trianle{float3{314.0, 0.0, 456.0}, float3{314.0, 330.0, 456.0}, float3{265.0, 330.0, 296.0}, float3{0.956166, 0, -0.292826}, false, BRDF_rate},
    Trianle{float3{314.0, 0.0, 456.0}, float3{265.0, 0.0, 296.0}, float3{265.0, 330.0, 296.0}, float3{0.956166, 0, -0.292826}, false, BRDF_rate},
    
    // Front
    Trianle{float3{265.0, 0.0, 296.0}, float3{265.0, 330.0, 296.0}, float3{423.0, 330.0, 247.0}, float3{-0.296209, 0, -0.955123}, false, BRDF_rate},
    Trianle{float3{265.0, 0.0, 296.0}, float3{423.0, 0.0, 247.0}, float3{423.0, 330.0, 247.0}, float3{-0.296209, 0, -0.955123}, false, BRDF_rate}
};

__constant__ Trianle d_scene_objects[OBJ_TRI_COUNT];
// camera position
__constant__ float3 d_camera_position = float3{278, 273, -800};
__constant__ float3 d_camera_direction = float3{0, 0, 1};
__constant__ float3 d_camera_up_direction = float3{0, 1, 0};
__constant__ float3 d_camera_left_direction = float3{-1, 0, 0};

__constant__ float d_camera_focal_length = 3.5;
__constant__ float d_camera_width = 2.5;
__constant__ float d_camera_height = 2.5;
__constant__ float d_camera_pixel_width = 2.5 / RENDER_WIDTH;
__constant__ float d_camera_pixel_height= 2.5 / RENDER_HEIGHT;


__device__ inline float mixed_product(float3 vec_a, float3 vec_b, float3 vec_c)
{
    return vec_a.x * (vec_b.y * vec_c.z - vec_b.z * vec_c.y) + 
        vec_a.y * (vec_b.z * vec_c.x - vec_b.x * vec_c.z) + 
        vec_a.z * (vec_b.x * vec_c.y - vec_b.y * vec_c.x);
}


__device__ inline float3 sub_float3(float3 opr1, float3 opr2)
{
    return make_float3(opr1.x - opr2.x, opr1.y - opr2.y, opr1.z - opr2.z);
}


__device__ inline float3 scalar_mult_float3(float3 vec, float scalar)
{
    return make_float3(vec.x * scalar, vec.y * scalar, vec.z * scalar);
}

__device__ float dot(float3 opr1, float3 opr2)
{
    return opr1.x * opr2.x + opr1.y * opr2.y + opr1.z * opr2.z;
}

__device__ inline float3 add_float3(float3 opr1, float3 opr2)
{
    return make_float3(opr1.x + opr2.x, opr1.y + opr2.y, opr1.z + opr2.z);
}


__device__ float size(Trianle triangle)
{
    float3 vec1 = sub_float3(triangle.tri_b, triangle.tri_a);
    float3 vec2 = sub_float3(triangle.tri_c, triangle.tri_a);
    float3 cross_product = make_float3(vec1.y * vec2.z - vec1.z * vec2.y, vec1.z * vec2.x - vec1.x * vec2.z, vec1.x * vec2.y - vec1.y * vec2.x);
    return 0.5 * norm3df(cross_product.x, cross_product.y, cross_product.z);
}


__device__ float3 check_obj_hit(int src_tri_idx, float3 src_point, float3 direction, int& hit_obj_idx)
{
    // normalize direction
    float div_length = 1 / norm3df(direction.x, direction.y, direction.z);
    float3 normal_direction = make_float3(direction.x * div_length, direction.y * div_length, direction.z * div_length);

    hit_obj_idx = -1;

    float3 hit_point;
    float min_distance = 2147483647;

    for (int i = 0; i < OBJ_TRI_COUNT; ++i) {
        if (i == src_tri_idx) {
            continue;
        }
        // make shadow
        Trianle shadow_tri = Trianle{sub_float3(d_scene_objects[i].tri_a, scalar_mult_float3(normal_direction, dot(normal_direction, sub_float3(d_scene_objects[i].tri_a, src_point)))),
            sub_float3(d_scene_objects[i].tri_b, scalar_mult_float3(normal_direction, dot(normal_direction, sub_float3(d_scene_objects[i].tri_b, src_point)))),
            sub_float3(d_scene_objects[i].tri_c, scalar_mult_float3(normal_direction, dot(normal_direction, sub_float3(d_scene_objects[i].tri_c, src_point)))),
            normal_direction};

        // check in center
        float3 vec_pa = sub_float3(shadow_tri.tri_a, src_point);
        float3 vec_pb = sub_float3(shadow_tri.tri_b, src_point);
        float3 vec_pc = sub_float3(shadow_tri.tri_c, src_point);

        float papb = mixed_product(normal_direction, vec_pa, vec_pb);
        float pbpc = mixed_product(normal_direction, vec_pb, vec_pc);
        float pcpa = mixed_product(normal_direction, vec_pc, vec_pa);
        if ((papb > 0 && pbpc > 0 && pcpa > 0) || (papb < 0 && pbpc < 0 && pcpa < 0)) {
            // in center
            // get hit point
            // get coordinary, reuse vec_pb ,vec_pc
            vec_pb = sub_float3(shadow_tri.tri_b, shadow_tri.tri_a);
            vec_pc = sub_float3(shadow_tri.tri_c, shadow_tri.tri_a);
            vec_pa = sub_float3(src_point, shadow_tri.tri_a);
            float divider = vec_pb.x * vec_pc.y - vec_pb.y * vec_pc.x;
            float rate_a = (vec_pc.y * vec_pa.x - vec_pc.x * vec_pa.y) / divider;
            float rate_b = (-vec_pb.y * vec_pa.x + vec_pb.x * vec_pa.y) / divider;

            vec_pb = sub_float3(d_scene_objects[i].tri_b, d_scene_objects[i].tri_a);
            vec_pc = sub_float3(d_scene_objects[i].tri_c, d_scene_objects[i].tri_a);
            vec_pa.x = d_scene_objects[i].tri_a.x + rate_a * vec_pb.x + rate_b * vec_pc.x;
            vec_pa.y = d_scene_objects[i].tri_a.y + rate_a * vec_pb.y + rate_b * vec_pc.y;
            vec_pa.z = d_scene_objects[i].tri_a.z + rate_a * vec_pb.z + rate_b * vec_pc.z;

            float distance = dot(sub_float3(vec_pa, src_point), normal_direction);
            // printf("Rate : %f %f %f\n", rate_a, rate_b, distance / norm3df(vec_pa.x - src_point.x, vec_pa.y - src_point.y, vec_pa.z - src_point.z));
            if (distance > 0) {
                // printf("In Center : %f, %f, %f %f\n", papb, pbpc, pcpa, distance);
                // ray will hit object
                if (distance < min_distance) {
                    min_distance = distance;
                    hit_point = vec_pa;
                    hit_obj_idx = i;
                }
            }
        }
    }

    // printf("Src : %d   Dst : %d   Direction : %f, %f, %f\n", src_tri_idx, hit_obj_idx, direction.x, direction.y, direction.z);
    return hit_point;
}



__device__ float3 check_light_hit(int src_tri_idx, float3 src_point, float3 direction, int& hit_obj_idx)
{
    float3 hit_point = check_obj_hit(src_tri_idx, src_point, direction, hit_obj_idx);
    if (hit_obj_idx > -1 && !d_scene_objects[hit_obj_idx].is_light) {
        hit_obj_idx = -1;
    }

    return hit_point;
}

/*
__device__ float shade_recurse(int object_idx, float3 src_point, float3 direction, curandState* curand_state)
{
    // Contribution from the light source.
    float l_dir = 0;
    for (int i = 0; i < LIGHT_TRI_COUNT; ++i) {
        // random select a point on light triangle
        float rand_x = curand_uniform(curand_state);
        float rand_y = curand_uniform(curand_state);
        if (rand_x + rand_y > 1) {
            rand_x = 1 - rand_x;
            rand_y = 1 - rand_y;
        }
        float3 random_point = add_float3(d_scene_objects[i].tri_a, add_float3(scalar_mult_float3(sub_float3(d_scene_objects[i].tri_b, d_scene_objects[i].tri_a), rand_x), scalar_mult_float3(sub_float3(d_scene_objects[i].tri_c, d_scene_objects[i].tri_a), rand_y)));

        // test block
        float3 obj_light_direction = sub_float3(random_point, src_point);
        int test_block_idx;
        check_obj_hit(-1, src_point, obj_light_direction, test_block_idx);
        // printf("Direction %f %f %f %d\n", obj_light_direction.x, obj_light_direction.y, obj_light_direction.z, test_block_idx);
        if (test_block_idx == i) {
            // printf("Hit Light!\n");
            float direction_length_square = obj_light_direction.x * obj_light_direction.x + obj_light_direction.y * obj_light_direction.y + obj_light_direction.z * obj_light_direction.z;
            l_dir += d_light_irradiance * BRDF_rate * dot(d_scene_objects[object_idx].normal_line, obj_light_direction) * -1 * dot(d_scene_objects[i].normal_line, obj_light_direction) 
                        / direction_length_square / direction_length_square * size(d_scene_objects[i]);
            // printf("Shade %d %f %f\n", i, dot(d_light_triangle[i].normal_line, obj_light_direction), l_dir);
        }
    }

    return l_dir;
    // Contribution from other reflectors.
    float l_indir = 0;

    // test Russian Roulette
    float rr_result = curand_uniform(curand_state);
    if (rr_result < RR_RATE) {
        // random select a ray from src_point
        float cosine_theta = 2 * (curand_uniform(curand_state) - 0.5);
        float sine_theta = sqrtf(1 - cosine_theta * cosine_theta);
        float fai_value = 2 * PI * curand_uniform(curand_state);
        float3 ray_direction = make_float3(sine_theta * cosf(fai_value), sine_theta * sinf(fai_value), cosine_theta);
        if (dot(ray_direction, d_scene_objects[object_idx].normal_line) < 0) {
            ray_direction.x *= -1;
            ray_direction.y *= -1;
            ray_direction.z *= -1;
            cosine_theta *= -1;
        }

        int hit_obj_idx;
        float3 hit_point = check_obj_hit(object_idx, src_point, ray_direction, hit_obj_idx);
        if (hit_obj_idx > -1 && !d_scene_objects[hit_obj_idx].is_light) {
            // printf("Hit Object!\n");
            ray_direction.x *= -1;
            ray_direction.y *= -1;
            ray_direction.z *= -1;
            l_indir = shade(hit_obj_idx, hit_point, ray_direction, curand_state) * BRDF_rate * dot(ray_direction, d_scene_objects[hit_obj_idx].normal_line) * 2 * PI / RR_RATE;
        }
    }

    // printf("Shade %f\n", l_dir + l_indir);
    return l_dir + l_indir;
}
*/
__device__ float stack_dir[SHARED_MEM_CAP];
__device__ float stack_indir_rate[SHARED_MEM_CAP];

__device__ float shade(int object_idx, float3 src_point, float3 direction, hiprandState* curand_state)
{
    // __shared__ float stack_dir[SHARED_MEM_CAP];
    // __shared__ float stack_indir_rate[SHARED_MEM_CAP];

    // int stack_size = 0;
    float l_dir = 0;
    int stack_offset = ((blockIdx.y * TILE_SIZE + threadIdx.y) * RENDER_WIDTH + (blockIdx.x * TILE_SIZE + threadIdx.x)) * STACK_CAPACITY;
    int stack_ori = stack_offset;
    float3 out_direction = direction; // use in BRDF, here is ignored.
    float3 ray_src = src_point;
    int src_object_idx = object_idx;
    while (true) {
        // Contribution from the light source.
        l_dir = 0;
        for (int i = 0; i < LIGHT_TRI_COUNT; ++i) {
            // random select a point on light triangle
            float rand_x = hiprand_uniform(curand_state);
            float rand_y = hiprand_uniform(curand_state);
            if (rand_x + rand_y > 1) {
                rand_x = 1 - rand_x;
                rand_y = 1 - rand_y;
            }
            float3 random_point = add_float3(d_scene_objects[i].tri_a, add_float3(scalar_mult_float3(sub_float3(d_scene_objects[i].tri_b, d_scene_objects[i].tri_a), rand_x), scalar_mult_float3(sub_float3(d_scene_objects[i].tri_c, d_scene_objects[i].tri_a), rand_y)));
    
            // test block
            float3 obj_light_direction = sub_float3(random_point, ray_src);
            int test_block_idx;
            check_obj_hit(-1, ray_src, obj_light_direction, test_block_idx);
            // printf("Direction %f %f %f %d\n", obj_light_direction.x, obj_light_direction.y, obj_light_direction.z, test_block_idx);
            if (test_block_idx == i) {
                // printf("Hit Light!\n");
                float direction_length_square = obj_light_direction.x * obj_light_direction.x + obj_light_direction.y * obj_light_direction.y + obj_light_direction.z * obj_light_direction.z;
                l_dir += d_light_irradiance * d_scene_objects[src_object_idx].brdf_rate * dot(d_scene_objects[src_object_idx].normal_line, obj_light_direction) * -1 * dot(d_scene_objects[i].normal_line, obj_light_direction) 
                            / direction_length_square / direction_length_square * size(d_scene_objects[i]);
                // printf("Shade %d %f %f\n", i, dot(d_light_triangle[i].normal_line, obj_light_direction), l_dir);
            }
        }

        // Contribution from other reflectors.
        // test Russian Roulette
        float rr_result = hiprand_uniform(curand_state);
        if (rr_result < RR_RATE) {
            float indir_rate = 0;
            // random select a ray from src_point
            float cosine_theta = 2 * (hiprand_uniform(curand_state) - 0.5);
            float sine_theta = sqrtf(1 - cosine_theta * cosine_theta);
            float fai_value = 2 * PI * hiprand_uniform(curand_state);
            float3 ray_direction = make_float3(sine_theta * cosf(fai_value), sine_theta * sinf(fai_value), cosine_theta);
            if (dot(ray_direction, d_scene_objects[src_object_idx].normal_line) < 0) {
                ray_direction.x *= -1;
                ray_direction.y *= -1;
                ray_direction.z *= -1;
                cosine_theta *= -1;
            }

            int hit_obj_idx;
            float3 hit_point = check_obj_hit(src_object_idx, ray_src, ray_direction, hit_obj_idx);
            if (hit_obj_idx > -1 && !d_scene_objects[hit_obj_idx].is_light) {
                // printf("Hit Object!\n");
                ray_direction.x *= -1;
                ray_direction.y *= -1;
                ray_direction.z *= -1;
                indir_rate = d_scene_objects[hit_obj_idx].brdf_rate * dot(ray_direction, d_scene_objects[hit_obj_idx].normal_line) / RR_RATE;
                src_object_idx = hit_obj_idx;
                ray_src = hit_point;
                out_direction = ray_direction;

                stack_dir[stack_offset] = l_dir;
                stack_indir_rate[stack_offset] = indir_rate;
                ++stack_offset;
            }
            else {
                // stack_dir[stack_offset] = l_dir;
                // stack_indir_rate[stack_offset] = indir_rate;
                // ++stack_offset;
                break;
            }
        }
        else {
            break;
        }
    }

    // calc final irradiance
    for (int i = stack_offset - 1; i >= stack_ori; --i) {
        // printf("%f %f\n", stack_indir_rate[i], stack_dir[i]);
        l_dir *= stack_indir_rate[i];
        l_dir += stack_dir[i];
    }
    return l_dir;
}

__device__ __forceinline__ float ray_generation(float3 pixel_center_position, hiprandState* curand_states)
{
    float pixel_radiance = 0;
    for (int i = 0; i < SPP; ++i) {
        float width_bias = d_camera_pixel_width * (hiprand_uniform(&curand_states[threadIdx.x]) - 0.5);
        float height_bias = d_camera_pixel_height * (hiprand_uniform(&curand_states[threadIdx.x]) - 0.5);
        int hit_obj_idx;
        // printf("Pixel bias : %f %f\n", width_bias, height_bias);


        float3 ray_direction = sub_float3(add_float3(pixel_center_position, make_float3(width_bias, 0, height_bias)), d_camera_position);
        float3 hit_light_point = check_light_hit(-1, d_camera_position, ray_direction, hit_obj_idx);
        if (hit_obj_idx > -1) {
            // printf("Ray Hit!\n");
            pixel_radiance += 1.0 / SPP * d_light_irradiance;
        }
        else {
            float3 hit_point = check_obj_hit(-1, d_camera_position, ray_direction, hit_obj_idx);
            if (hit_obj_idx > -1) {
                // printf("Obj Hit!\n");
                float3 reverse_ray_direction = make_float3(-ray_direction.x, -ray_direction.y, -ray_direction.z);
                pixel_radiance += 1.0 / SPP * shade(hit_obj_idx, hit_point, reverse_ray_direction, &curand_states[threadIdx.x]);
                // printf("Ray Obj General : %f\n", pixel_radiance);
            }
        }
    }

    // printf("Ray General : %f\n", 1.0 / SPP * d_light_irradiance);
    return pixel_radiance;
}



__global__ void render_pixel(unsigned char* target_img, hiprandState* curand_states)
{
    int target_pixel_width = blockIdx.x * TILE_SIZE + threadIdx.x;
    int target_pixel_height = blockIdx.y * TILE_SIZE + threadIdx.y;
    // printf("%d, %d\n", target_pixel_width, target_pixel_height);

    // printf("%f %f %f\n", d_camera_position.x, d_camera_position.y, d_camera_position.z);

    float3 delta_left = scalar_mult_float3(d_camera_left_direction, (target_pixel_width + 0.5 - RENDER_WIDTH / 2.0) * d_camera_pixel_width);
    float3 delta_up = scalar_mult_float3(d_camera_up_direction, (target_pixel_height + 0.5 - RENDER_HEIGHT / 2.0) * d_camera_pixel_height);
    float3 delta = add_float3(delta_left, add_float3(delta_up, scalar_mult_float3(d_camera_direction, d_camera_focal_length)));
    // float3 delta = make_float3((target_pixel_width + 0.5 - RENDER_WIDTH / 2.0) * d_camera_pixel_width, d_camera_focal_length, (target_pixel_height + 0.5 - RENDER_HEIGHT / 2.0) * d_camera_pixel_height);
    float3 pixel_center = make_float3(d_camera_position.x + delta.x, d_camera_position.y + delta.y, d_camera_position.z + delta.z);
    float pixel_radiance = ray_generation(pixel_center, curand_states);
    // float pixel_radiance = d_light_irradiance * curand_uniform(&curand_states[threadIdx.x]);

    // Gamma correction
    pixel_radiance /= d_light_irradiance;
    if (pixel_radiance > 1) {
        pixel_radiance = 1;
    }
    pixel_radiance = powf(pixel_radiance, 0.454545454545);

    
    unsigned char rgb_value = (unsigned char)(pixel_radiance * 255);
    // printf("%d, %d : %d\n", target_pixel_width, target_pixel_height, rgb_value);
    int base_idx = 3 * (target_pixel_height * RENDER_WIDTH + target_pixel_width);
    target_img[base_idx] = rgb_value;
    target_img[base_idx + 1] = rgb_value;
    target_img[base_idx + 2] = rgb_value;
}


__global__ void init_curand(hiprandState* curand_states, int seed)
{
    hiprand_init(seed, threadIdx.x, 0, &(curand_states[threadIdx.x]));
}

int main()
{
    dim3 grid{RENDER_WIDTH / TILE_SIZE, RENDER_HEIGHT / TILE_SIZE, 1};
    dim3 block{TILE_SIZE, TILE_SIZE, 1};

    unsigned char* d_target_img;
    hipMalloc(&d_target_img, RENDER_WIDTH * RENDER_HEIGHT * 3);

    hiprandState* curand_states;
    hipMalloc(&curand_states, TILE_SIZE * sizeof(hiprandState));

    init_curand <<<1, TILE_SIZE>>> (curand_states, 0);
    hipDeviceSynchronize();
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hiprand init launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // cudaMemcpyToSymbol(d_light_triangle, h_light_triangle, sizeof(Trianle) * LIGHT_TRI_COUNT, cudaMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_scene_objects), h_scene_objects, sizeof(h_scene_objects));

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "before render launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    render_pixel <<<grid, block>>> (d_target_img, curand_states);
    
    unsigned char* h_target_img = (unsigned char*)malloc(RENDER_WIDTH * RENDER_HEIGHT * 3);

    hipDeviceSynchronize();

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "render launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    
    hipMemcpy(h_target_img, d_target_img, RENDER_WIDTH * RENDER_HEIGHT * 3, hipMemcpyDeviceToHost);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "copy launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    save_image(h_target_img, RENDER_WIDTH, RENDER_HEIGHT);
    free(h_target_img);

    hipFree(d_target_img);
    hipFree(curand_states);
    hipDeviceReset();

    return 0;
}