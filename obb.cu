#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#ifdef LARGE
#define RENDER_WIDTH 1024
#define RENDER_HEIGHT 1024
#else
#define RENDER_WIDTH 128
#define RENDER_HEIGHT 128
#endif

#define TILE_SIZE 16
#define STACK_CAPACITY 128
#define SHARED_MEM_CAP STACK_CAPACITY * RENDER_WIDTH * RENDER_HEIGHT
#define SPP 1024
#define RR_RATE 0.9
#define PI 3.1415926


// BMP Operation
// 文件信息头结构体
typedef struct
{
    unsigned int   bfSize;        // 文件大小 以字节为单位(2-5字节)
    unsigned short bfReserved1;   // 保留，必须设置为0 (6-7字节)
    unsigned short bfReserved2;   // 保留，必须设置为0 (8-9字节)
    unsigned int   bfOffBits;     // 从文件头到像素数据的偏移  (10-13字节)
} _BITMAPFILEHEADER;

//图像信息头结构体
typedef struct
{
    unsigned int    biSize;          // 此结构体的大小 (14-17字节)
    int             biWidth;         // 图像的宽  (18-21字节)
    int             biHeight;        // 图像的高  (22-25字节)
    unsigned short  biPlanes;        // 表示bmp图片的平面属，显然显示器只有一个平面，所以恒等于1 (26-27字节)
    unsigned short  biBitCount;      // 一像素所占的位数，一般为24   (28-29字节)
    unsigned int    biCompression;   // 说明图象数据压缩的类型，0为不压缩。 (30-33字节)
    unsigned int    biSizeImage;     // 像素数据所占大小, 这个值应该等于上面文件头结构中bfSize-bfOffBits (34-37字节)
    int             biXPelsPerMeter; // 说明水平分辨率，用象素/米表示。一般为0 (38-41字节)
    int             biYPelsPerMeter; // 说明垂直分辨率，用象素/米表示。一般为0 (42-45字节)
    unsigned int    biClrUsed;       // 说明位图实际使用的彩色表中的颜色索引数（设为0的话，则说明使用所有调色板项）。 (46-49字节)
    unsigned int    biClrImportant;  // 说明对图象显示有重要影响的颜色索引的数目，如果是0，表示都重要。(50-53字节)
} _BITMAPINFOHEADER;

__host__ void save_image(unsigned char* target_img, int width, int height)
{
    FILE* file_ptr = fopen("RenderResult.bmp", "wb+");

    unsigned short fileType = 0x4d42;
    _BITMAPFILEHEADER fileHeader;
    _BITMAPINFOHEADER infoHeader;

    fileHeader.bfSize = (width) * (height) * 3 + 54;
    fileHeader.bfReserved1 = 0;
    fileHeader.bfReserved2 = 0;
    fileHeader.bfOffBits = 54;

    infoHeader.biSize = 40;
    infoHeader.biWidth = width;
    infoHeader.biHeight = height;
    infoHeader.biPlanes = 1;
    infoHeader.biBitCount = 24;
    infoHeader.biCompression = 0;
    infoHeader.biSizeImage = (width) * (height) * 3;
    infoHeader.biXPelsPerMeter = 0;
    infoHeader.biYPelsPerMeter = 0;
    infoHeader.biClrUsed = 0;
    infoHeader.biClrImportant = 0;

    fwrite(&fileType, sizeof(unsigned short), 1, file_ptr);
    fwrite(&fileHeader, sizeof(_BITMAPFILEHEADER), 1, file_ptr);
    fwrite(&infoHeader, sizeof(_BITMAPINFOHEADER), 1, file_ptr);

    fwrite(target_img, sizeof(unsigned char), (height) * (width) * 3, file_ptr);

    fclose(file_ptr);
}


// 3D resources

struct Trianle {
    float3 tri_a;
    float3 tri_b;
    float3 tri_c;
    float3 normal_line;
    bool is_light;
    float brdf_rate;
};

#define LIGHT_TRI_COUNT 2
__constant__ float d_light_irradiance = 42;

#define BRDF_rate 0.5
__constant__ int OBJ_TRI_COUNT;

// Trianle{float3{}, float3{}, float3{}, float3{}, false, BRDF_rate},
// Trianle* h_scene_objects;
__device__ Trianle* d_scene_objects;
// camera position
__constant__ float3 d_camera_position = float3{150, -400, 150};
__constant__ float3 d_camera_direction = float3{0, 1, 0};
__constant__ float3 d_camera_up_direction = float3{0, 0, 1};
__constant__ float3 d_camera_left_direction = float3{1, 0, 0};
// 浮点精度考虑，设置较大焦距和成像平面
__constant__ float d_camera_focal_length = 200;
__constant__ float d_camera_width = 150;
__constant__ float d_camera_height = 150;
__constant__ float d_camera_pixel_width = 150.0 / RENDER_WIDTH;
__constant__ float d_camera_pixel_height= 150.0 / RENDER_HEIGHT;

__host__ void load_obb_file(string file_path)
{
    ifstream fin("test.off");

    string head;
    fin >> head;
    int point_count, surface_count, temp;
    fin >> point_count >> surface_count >> temp;

    float3* points = new float3[point_count];
    float min_x = 214748364;
    float min_y = 214748364;
    float min_z = 214748364;
    float max_x = -214748368;
    float max_y = -214748368;
    float max_z = -214748368;
    for (int i = 0; i < point_count; ++i) {
        float x, y, z;
        fin >> x >> y >> z;
        if (min_x > x) {
            min_x = x;
        }
        if (min_y > y) {
            min_y = y;
        }
        if (min_z > z) {
            min_z = z;
        }
        if (max_x < x) {
            max_x = x;
        }
        if (max_y < y) {
            max_y = y;
        }
        if (max_z < z) {
            max_z = z;
        }
        points[i] = make_float3(x, y, z);
    }

    printf("%f %f %f %f %f %f\n", min_x, min_y, min_z, max_x, max_y, max_z);
    float x_scale = 250 / (max_x - min_x);
    float y_scale = 250 / (max_y - min_y);
    float z_scale = 250 / (max_z - min_z);

    float scale = 2147483647;
    if (x_scale < scale) {
        scale = x_scale;
    }
    if (y_scale < scale) {
        scale = y_scale;
    }
    if (z_scale < scale) {
        scale = z_scale;
    }

    // printf("%f\n", scale);
    for (int i = 0; i < point_count; ++i) {
        points[i].x -= min_x;
        points[i].x *= scale;
        points[i].x += 25;

        points[i].y -= min_y;
        points[i].y *= scale;
        points[i].y += 25;

        points[i].z -= min_z;
        points[i].z *= scale;
        points[i].z += 25;
    }

    Trianle* h_scene_objects = new Trianle[surface_count + 4];
    // Light
    h_scene_objects[0] = Trianle{float3{110, 110, 300}, float3{110, 190, 300}, float3{190, 110, 300}, float3{0, 0, -1}, true, BRDF_rate};
    h_scene_objects[1] = Trianle{float3{190, 110, 300}, float3{110, 190, 300}, float3{190, 190, 300}, float3{0, 0, -1}, true, BRDF_rate},
    // Ceiling
    h_scene_objects[2] = Trianle{float3{0, 0, 0}, float3{0, 300, 0}, float3{300, 0, 0}, float3{0, 0, 1}, false, BRDF_rate};
    h_scene_objects[3] = Trianle{float3{0, 300, 0}, float3{300, 0, 0}, float3{300, 300, 0}, float3{0, 0, 1}, false, BRDF_rate};
    for (int i = 4; i < surface_count + 4; ++i) {
        int index_x, index_y, index_z;
        fin >> temp >> index_x >> index_y >> index_z;
        float3 yx = make_float3(points[index_y].x - points[index_x].x, points[index_y].y - points[index_x].y, points[index_y].z - points[index_x].z);
        float3 yz = make_float3(points[index_y].x - points[index_z].x, points[index_y].y - points[index_z].y, points[index_y].z - points[index_z].z);
        float3 normal_line = make_float3(yx.y * yz.z - yx.z * yz.y, yx.z * yz.x - yx.x * yz.z, yx.x * yz.y - yx.y * yz.x);
        float normal_length = 1 / sqrt(normal_line.x * normal_line.x + normal_line.y * normal_line.y + normal_line.z * normal_line.z);
        h_scene_objects[i] = Trianle{points[index_x], points[index_y], points[index_z], float3{normal_length * normal_line.x, normal_length * normal_line.y, normal_length * normal_line.z}, false, BRDF_rate};
        // printf("%f, %f, %f\n", h_scene_objects[i].normal_line.x, h_scene_objects[i].normal_line.y, h_scene_objects[i].normal_line.z); 
    }

    fin.close();

    surface_count += 4;
    hipMemcpyToSymbol(HIP_SYMBOL(OBJ_TRI_COUNT), &surface_count, sizeof(int));
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "copy int launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    Trianle* temp_scene_objects;
    hipMalloc(&temp_scene_objects, sizeof(Trianle) * surface_count);
    hipMemcpy(temp_scene_objects, h_scene_objects, sizeof(Trianle) * surface_count, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_scene_objects), &temp_scene_objects, sizeof(Trianle*));
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "copy array launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    delete[] points;
    delete[] h_scene_objects;

    printf("Load Done.\n");
}

__device__ inline float mixed_product(float3 vec_a, float3 vec_b, float3 vec_c)
{
    return vec_a.x * (vec_b.y * vec_c.z - vec_b.z * vec_c.y) + 
        vec_a.y * (vec_b.z * vec_c.x - vec_b.x * vec_c.z) + 
        vec_a.z * (vec_b.x * vec_c.y - vec_b.y * vec_c.x);
}


__device__ inline float3 sub_float3(float3 opr1, float3 opr2)
{
    return make_float3(opr1.x - opr2.x, opr1.y - opr2.y, opr1.z - opr2.z);
}


__device__ inline float3 scalar_mult_float3(float3 vec, float scalar)
{
    return make_float3(vec.x * scalar, vec.y * scalar, vec.z * scalar);
}

__device__ float dot(float3 opr1, float3 opr2)
{
    return opr1.x * opr2.x + opr1.y * opr2.y + opr1.z * opr2.z;
}

__device__ inline float3 add_float3(float3 opr1, float3 opr2)
{
    return make_float3(opr1.x + opr2.x, opr1.y + opr2.y, opr1.z + opr2.z);
}


__device__ float size(Trianle triangle)
{
    float3 vec1 = sub_float3(triangle.tri_b, triangle.tri_a);
    float3 vec2 = sub_float3(triangle.tri_c, triangle.tri_a);
    float3 cross_product = make_float3(vec1.y * vec2.z - vec1.z * vec2.y, vec1.z * vec2.x - vec1.x * vec2.z, vec1.x * vec2.y - vec1.y * vec2.x);
    return 0.5 * norm3df(cross_product.x, cross_product.y, cross_product.z);
}


__device__ float3 check_obj_hit(int src_tri_idx, float3 src_point, float3 direction, int& hit_obj_idx)
{
    // normalize direction
    float div_length = 1 / norm3df(direction.x, direction.y, direction.z);
    float3 normal_direction = make_float3(direction.x * div_length, direction.y * div_length, direction.z * div_length);

    hit_obj_idx = -1;

    float3 hit_point;
    float min_distance = 2147483647;

    for (int i = 0; i < OBJ_TRI_COUNT; ++i) {
        if (i == src_tri_idx) {
            continue;
        }
        // make shadow
        Trianle shadow_tri = Trianle{sub_float3(d_scene_objects[i].tri_a, scalar_mult_float3(normal_direction, dot(normal_direction, sub_float3(d_scene_objects[i].tri_a, src_point)))),
            sub_float3(d_scene_objects[i].tri_b, scalar_mult_float3(normal_direction, dot(normal_direction, sub_float3(d_scene_objects[i].tri_b, src_point)))),
            sub_float3(d_scene_objects[i].tri_c, scalar_mult_float3(normal_direction, dot(normal_direction, sub_float3(d_scene_objects[i].tri_c, src_point)))),
            normal_direction};

        // check in center
        float3 vec_pa = sub_float3(shadow_tri.tri_a, src_point);
        float3 vec_pb = sub_float3(shadow_tri.tri_b, src_point);
        float3 vec_pc = sub_float3(shadow_tri.tri_c, src_point);

        float papb = mixed_product(normal_direction, vec_pa, vec_pb);
        float pbpc = mixed_product(normal_direction, vec_pb, vec_pc);
        float pcpa = mixed_product(normal_direction, vec_pc, vec_pa);
        if ((papb > 0 && pbpc > 0 && pcpa > 0) || (papb < 0 && pbpc < 0 && pcpa < 0)) {
            // in center
            // get hit point
            // get coordinary, reuse vec_pb ,vec_pc
            vec_pb = sub_float3(shadow_tri.tri_b, shadow_tri.tri_a);
            vec_pc = sub_float3(shadow_tri.tri_c, shadow_tri.tri_a);
            vec_pa = sub_float3(src_point, shadow_tri.tri_a);
            float divider = vec_pb.x * vec_pc.y - vec_pb.y * vec_pc.x;
            float rate_a = (vec_pc.y * vec_pa.x - vec_pc.x * vec_pa.y) / divider;
            float rate_b = (-vec_pb.y * vec_pa.x + vec_pb.x * vec_pa.y) / divider;

            vec_pb = sub_float3(d_scene_objects[i].tri_b, d_scene_objects[i].tri_a);
            vec_pc = sub_float3(d_scene_objects[i].tri_c, d_scene_objects[i].tri_a);
            vec_pa.x = d_scene_objects[i].tri_a.x + rate_a * vec_pb.x + rate_b * vec_pc.x;
            vec_pa.y = d_scene_objects[i].tri_a.y + rate_a * vec_pb.y + rate_b * vec_pc.y;
            vec_pa.z = d_scene_objects[i].tri_a.z + rate_a * vec_pb.z + rate_b * vec_pc.z;

            float distance = dot(sub_float3(vec_pa, src_point), normal_direction);
            // printf("Rate : %f %f %f\n", rate_a, rate_b, distance / norm3df(vec_pa.x - src_point.x, vec_pa.y - src_point.y, vec_pa.z - src_point.z));
            if (distance > 0) {
                // printf("In Center : %f, %f, %f %f\n", papb, pbpc, pcpa, distance);
                // ray will hit object
                if (distance < min_distance) {
                    min_distance = distance;
                    hit_point = vec_pa;
                    hit_obj_idx = i;
                }
            }
        }
    }

    // printf("Src : %d   Dst : %d   Direction : %f, %f, %f\n", src_tri_idx, hit_obj_idx, direction.x, direction.y, direction.z);
    return hit_point;
}



__device__ float3 check_light_hit(int src_tri_idx, float3 src_point, float3 direction, int& hit_obj_idx)
{
    float3 hit_point = check_obj_hit(src_tri_idx, src_point, direction, hit_obj_idx);
    if (hit_obj_idx > -1 && !d_scene_objects[hit_obj_idx].is_light) {
        hit_obj_idx = -1;
    }

    return hit_point;
}

__device__ float stack_dir[SHARED_MEM_CAP];
__device__ float stack_indir_rate[SHARED_MEM_CAP];

__device__ float shade(int object_idx, float3 src_point, float3 direction, hiprandState* curand_state)
{
    // __shared__ float stack_dir[SHARED_MEM_CAP];
    // __shared__ float stack_indir_rate[SHARED_MEM_CAP];

    // int stack_size = 0;
    float l_dir = 0;
    int stack_offset = ((blockIdx.y * TILE_SIZE + threadIdx.y) * RENDER_WIDTH + (blockIdx.x * TILE_SIZE + threadIdx.x)) * STACK_CAPACITY;
    int stack_ori = stack_offset;
    float3 out_direction = direction; // use in BRDF, here is ignored.
    float3 ray_src = src_point;
    int src_object_idx = object_idx;
    while (true) {
        // Contribution from the light source.
        l_dir = 0;
        for (int i = 0; i < LIGHT_TRI_COUNT; ++i) {
            // random select a point on light triangle
            float rand_x = hiprand_uniform(curand_state);
            float rand_y = hiprand_uniform(curand_state);
            if (rand_x + rand_y > 1) {
                rand_x = 1 - rand_x;
                rand_y = 1 - rand_y;
            }
            float3 random_point = add_float3(d_scene_objects[i].tri_a, add_float3(scalar_mult_float3(sub_float3(d_scene_objects[i].tri_b, d_scene_objects[i].tri_a), rand_x), scalar_mult_float3(sub_float3(d_scene_objects[i].tri_c, d_scene_objects[i].tri_a), rand_y)));
    
            // test block
            float3 obj_light_direction = sub_float3(random_point, ray_src);
            int test_block_idx;
            check_obj_hit(-1, ray_src, obj_light_direction, test_block_idx);
            // printf("Direction %f %f %f %d\n", obj_light_direction.x, obj_light_direction.y, obj_light_direction.z, test_block_idx);
            if (test_block_idx == i) {
                // printf("Hit Light!\n");
                float direction_length_square = obj_light_direction.x * obj_light_direction.x + obj_light_direction.y * obj_light_direction.y + obj_light_direction.z * obj_light_direction.z;
                l_dir += d_light_irradiance * d_scene_objects[src_object_idx].brdf_rate * fabs(dot(d_scene_objects[src_object_idx].normal_line, obj_light_direction) * dot(d_scene_objects[i].normal_line, obj_light_direction)) 
                            / direction_length_square / direction_length_square * size(d_scene_objects[i]);
            }
        }

        // Contribution from other reflectors.
        // test Russian Roulette
        float rr_result = hiprand_uniform(curand_state);
        if (rr_result < RR_RATE) {
            float indir_rate = 0;
            // random select a ray from src_point
            float cosine_theta = 2 * (hiprand_uniform(curand_state) - 0.5);
            float sine_theta = sqrtf(1 - cosine_theta * cosine_theta);
            float fai_value = 2 * PI * hiprand_uniform(curand_state);
            float3 ray_direction = make_float3(sine_theta * cosf(fai_value), sine_theta * sinf(fai_value), cosine_theta);
            if (dot(ray_direction, d_scene_objects[src_object_idx].normal_line) * dot(out_direction, d_scene_objects[src_object_idx].normal_line) < 0) {
                ray_direction.x *= -1;
                ray_direction.y *= -1;
                ray_direction.z *= -1;
                cosine_theta *= -1;
            }

            int hit_obj_idx;
            float3 hit_point = check_obj_hit(src_object_idx, ray_src, ray_direction, hit_obj_idx);
            if (hit_obj_idx > -1 && !d_scene_objects[hit_obj_idx].is_light) {
                // printf("Hit Object!\n");
                ray_direction.x *= -1;
                ray_direction.y *= -1;
                ray_direction.z *= -1;
                indir_rate = d_scene_objects[hit_obj_idx].brdf_rate * fabs(dot(ray_direction, d_scene_objects[hit_obj_idx].normal_line)) / RR_RATE;
                src_object_idx = hit_obj_idx;
                ray_src = hit_point;
                out_direction = ray_direction;

                stack_dir[stack_offset] = l_dir;
                stack_indir_rate[stack_offset] = indir_rate;
                ++stack_offset;
            }
            else {
                // stack_dir[stack_offset] = l_dir;
                // stack_indir_rate[stack_offset] = indir_rate;
                // ++stack_offset;
                break;
            }
        }
        else {
            break;
        }
    }

    // calc final irradiance
    for (int i = stack_offset - 1; i >= stack_ori; --i) {
        // printf("%f %f\n", stack_indir_rate[i], stack_dir[i]);
        l_dir *= stack_indir_rate[i];
        l_dir += stack_dir[i];
    }
    return l_dir;
}

__device__ __forceinline__ float ray_generation(float3 pixel_center_position, hiprandState* curand_states)
{
    float pixel_radiance = 0;
    for (int i = 0; i < SPP; ++i) {
        float width_bias = d_camera_pixel_width * (hiprand_uniform(&curand_states[threadIdx.x]) - 0.5);
        float height_bias = d_camera_pixel_height * (hiprand_uniform(&curand_states[threadIdx.x]) - 0.5);
        int hit_obj_idx;
        // printf("Pixel bias : %f %f\n", width_bias, height_bias);


        float3 ray_direction = sub_float3(add_float3(pixel_center_position, make_float3(width_bias, 0, height_bias)), d_camera_position);
        float3 hit_light_point = check_light_hit(-1, d_camera_position, ray_direction, hit_obj_idx);
        if (hit_obj_idx > -1) {
            // printf("Ray Hit!\n");
            pixel_radiance += 1.0 / SPP * d_light_irradiance;
        }
        else {
            float3 hit_point = check_obj_hit(-1, d_camera_position, ray_direction, hit_obj_idx);
            if (hit_obj_idx > -1) {
                // printf("Obj Hit!\n");
                float3 reverse_ray_direction = make_float3(-ray_direction.x, -ray_direction.y, -ray_direction.z);
                pixel_radiance += 1.0 / SPP * shade(hit_obj_idx, hit_point, reverse_ray_direction, &curand_states[threadIdx.x]);
                // printf("Ray Obj General : %f\n", pixel_radiance);
            }
        }
    }

    // printf("Ray General : %f\n", 1.0 / SPP * d_light_irradiance);
    return pixel_radiance;
}



__global__ void render_pixel(unsigned char* target_img, hiprandState* curand_states)
{
    // printf("Hit\n");
    // printf("%f\n", d_scene_objects[0].tri_a.z);
    int target_pixel_width = blockIdx.x * TILE_SIZE + threadIdx.x;
    int target_pixel_height = blockIdx.y * TILE_SIZE + threadIdx.y;
    // printf("%d, %d\n", target_pixel_width, target_pixel_height);

    // printf("%f %f %f\n", d_camera_position.x, d_camera_position.y, d_camera_position.z);

    float3 delta_left = scalar_mult_float3(d_camera_left_direction, (target_pixel_width + 0.5 - RENDER_WIDTH / 2.0) * d_camera_pixel_width);
    float3 delta_up = scalar_mult_float3(d_camera_up_direction, (target_pixel_height + 0.5 - RENDER_HEIGHT / 2.0) * d_camera_pixel_height);
    float3 delta = add_float3(delta_left, add_float3(delta_up, scalar_mult_float3(d_camera_direction, d_camera_focal_length)));
    // float3 delta = make_float3((target_pixel_width + 0.5 - RENDER_WIDTH / 2.0) * d_camera_pixel_width, d_camera_focal_length, (target_pixel_height + 0.5 - RENDER_HEIGHT / 2.0) * d_camera_pixel_height);
    float3 pixel_center = make_float3(d_camera_position.x + delta.x, d_camera_position.y + delta.y, d_camera_position.z + delta.z);
    float pixel_radiance = ray_generation(pixel_center, curand_states);
    // float pixel_radiance = d_light_irradiance * curand_uniform(&curand_states[threadIdx.x]);

    // Gamma correction
    pixel_radiance /= d_light_irradiance;
    if (pixel_radiance > 1) {
        pixel_radiance = 1;
    }
    pixel_radiance = powf(pixel_radiance, 0.454545454545);

    
    unsigned char rgb_value = (unsigned char)(pixel_radiance * 255);
    // printf("%d, %d : %d\n", target_pixel_width, target_pixel_height, rgb_value);
    int base_idx = 3 * (target_pixel_height * RENDER_WIDTH + target_pixel_width);
    target_img[base_idx] = rgb_value;
    target_img[base_idx + 1] = rgb_value;
    target_img[base_idx + 2] = rgb_value;
}


__global__ void init_curand(hiprandState* curand_states, int seed)
{
    hiprand_init(seed, threadIdx.x, 0, &(curand_states[threadIdx.x]));
}

int main()
{
    string file_path;
    // cin >> file_path;

    dim3 grid{RENDER_WIDTH / TILE_SIZE, RENDER_HEIGHT / TILE_SIZE, 1};
    dim3 block{TILE_SIZE, TILE_SIZE, 1};

    unsigned char* d_target_img;
    hipMalloc(&d_target_img, RENDER_WIDTH * RENDER_HEIGHT * 3);

    hiprandState* curand_states;
    hipMalloc(&curand_states, TILE_SIZE * sizeof(hiprandState));

    init_curand <<<1, TILE_SIZE>>> (curand_states, 0);

    load_obb_file(file_path);

    hipDeviceSynchronize();
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hiprand init launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "before render launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    render_pixel <<<grid, block>>> (d_target_img, curand_states);
    
    unsigned char* h_target_img = (unsigned char*)malloc(RENDER_WIDTH * RENDER_HEIGHT * 3);

    hipDeviceSynchronize();

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "render launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    
    hipMemcpy(h_target_img, d_target_img, RENDER_WIDTH * RENDER_HEIGHT * 3, hipMemcpyDeviceToHost);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "copy launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    save_image(h_target_img, RENDER_WIDTH, RENDER_HEIGHT);
    free(h_target_img);

    hipFree(d_target_img);
    hipFree(curand_states);
    hipDeviceReset();
    return 0;
}